#include "hip/hip_runtime.h"
#include "CudaRuntime.cuh"

#include <cstdlib>
#include <thread>


namespace Cuda_Runtime
{
	__global__ void compute(const bool *primary_device_buffer, bool *secondary_device_buffer, int world_width)
	{
		int index = threadIdx.x;
		const int x_cord = index % world_width;
		const int y_cord = index / world_width;
		int alive_neighbours = 0;
		for(int i = 0; i < 9; i++)
		{
			for(int k = -1; k < 2; k++)
			{
				for(int j = -1; j < 2; j++)
				{
					if(j == 0 && k == 0)
					{
						continue;
					}
					if(primary_device_buffer[(y_cord + k) * world_width + (x_cord + j)] == true) alive_neighbours++;
				}
			}
		}
		if(alive_neighbours < 2 || alive_neighbours > 3)
		{
			secondary_device_buffer[index] = false;
		}
		else if(alive_neighbours == 3)
		{
			secondary_device_buffer[index] = true;
		}
		else
		{
			secondary_device_buffer[index] = primary_device_buffer[index];
		}
	}

	ComputeGPU::ComputeGPU(int world_width, bool *start_world_frame)
	{
		GPU_SYSTEM_STATE = INIT;
		this->world_width = world_width;
		this->host_world_buffer = start_world_frame;
	}

	void ComputeGPU::init()
	{
		buffer_size = world_width * world_width * sizeof(bool);
		//GPU device memory allocation
		hipMalloc(&primary_device_world_buffer, buffer_size);
		hipMalloc(&secondary_device_world_buffer, buffer_size);

		//Copy world buffer to GPU device
		hipMemcpy(primary_device_world_buffer, host_world_buffer, buffer_size,hipMemcpyHostToDevice);

		GPU_SYSTEM_STATE = RUNNING;
	}

	void ComputeGPU::run()
	{
		while(GPU_SYSTEM_STATE == RUNNING)
		{
			compute<<<1, world_width*world_width>>>(primary_device_world_buffer, secondary_device_world_buffer, world_width);
			hipMemcpy(host_world_buffer, secondary_device_world_buffer, buffer_size,hipMemcpyDeviceToHost);
			hipMemcpy(primary_device_world_buffer, secondary_device_world_buffer, buffer_size,hipMemcpyDeviceToDevice);
		}
	}

	void ComputeGPU::shutdown()
	{
		free(host_world_buffer);
		hipFree(primary_device_world_buffer);
		hipFree(secondary_device_world_buffer);
		GPU_SYSTEM_STATE = SHUTDOWN;
	}
}