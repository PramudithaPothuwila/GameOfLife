#include "hip/hip_runtime.h"
#include "CudaRuntime.cuh"

namespace Cuda_Runtime
{
	__global__ void compute(bool *primary_device_buffer, bool *secondary_device_buffer)
	{
		
	}

	ComputeGPU::ComputeGPU(int world_width, bool *start_world_frame)
	{
		this->world_width = world_width;
		this->host_world_buffer = start_world_frame;
	}

	void ComputeGPU::init()
	{
		const int buffer_size = world_width * world_width * sizeof(bool);
		//GPU device memory allocation
		hipMalloc(&primary_device_world_buffer, buffer_size);
		hipMalloc(&secondary_device_world_buffer, buffer_size);

		//Copy world buffer to GPU device
		hipMemcpy(primary_device_world_buffer, host_world_buffer, buffer_size,hipMemcpyHostToDevice);
	}
}