#include "hip/hip_runtime.h"
#include "cuda_runtime.cuh"
#include "system_state.h"

#include <cstdlib>
#include <iostream>
#include <thread>
#include "hip/hip_runtime.h"
#include ""


namespace cuda_runtime
 {
 	__global__ void compute(const bool *primary_device_buffer, bool *secondary_device_buffer, int world_width)
 	{
	    const int index = blockIdx.x * blockDim.x +threadIdx.x;
 		const int x_cord = index % world_width;
 		const int y_cord = index / world_width;
 		int alive_neighbours = 0;
        for (int k = -1; k < 2; k++)
        {
            for (int j = -1; j < 2; j++)
            {
                if (j == 0 && k == 0)
                {
                    continue;
                }
                if (primary_device_buffer[(y_cord + k) * world_width + (x_cord + j)] == true)
                {
                    alive_neighbours++;
                }
            }
        }
 		if(alive_neighbours < 2 || alive_neighbours > 3)
 		{
 			secondary_device_buffer[index] = false;
 		}
 		else if(alive_neighbours == 3)
 		{
 			secondary_device_buffer[index] = true;
 		}
 		else
 		{
 			secondary_device_buffer[index] = primary_device_buffer[index];
 		}
 	}

 	ComputeGpu::ComputeGpu(const int world_width, bool *start_world_frame)
 	{
 		Gpu_state_G = READY;
 		this->m_world_width = world_width;
 		this->m_host_world_buffer = start_world_frame;
 	}

 	void ComputeGpu::Init()
 	{
 		m_buffer_size = m_world_width * m_world_width * sizeof(bool);
 		//GPU device memory allocation
 		hipMalloc(&m_primary_device_world_buffer, m_buffer_size);
 		hipMalloc(&m_secondary_device_world_buffer, m_buffer_size);

 		//Copy world buffer to GPU device
 		hipMemcpy(m_primary_device_world_buffer, m_host_world_buffer, m_buffer_size,hipMemcpyHostToDevice);

        Gpu_state_G = RUNNING;
 	}
 	
 	void ComputeGpu::Run()
 	{
 		while(Gpu_state_G == RUNNING)
 		{
 			compute <<< m_world_width, m_world_width >>>(m_primary_device_world_buffer, m_secondary_device_world_buffer, m_world_width);
 			hipMemcpy(m_host_world_buffer, m_secondary_device_world_buffer, m_buffer_size,hipMemcpyDeviceToHost);
 			hipMemcpy(m_primary_device_world_buffer, m_secondary_device_world_buffer, m_buffer_size,hipMemcpyDeviceToDevice);
 			Grid_G->Update_World(m_host_world_buffer);
 			Grid_G->Print();
 		}
 	}

 	void ComputeGpu::Shutdown() const
    {
 		free(m_host_world_buffer);
 		hipFree(m_primary_device_world_buffer);
 		hipFree(m_secondary_device_world_buffer);
        Gpu_state_G = SHUTDOWN;
 	}
}	