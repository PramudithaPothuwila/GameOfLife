#include "hip/hip_runtime.h"
#include "CudaRuntime.cuh"

 #include <cstdlib>
 #include <iostream>
 #include <string>
 #include <thread>
#include "hip/hip_runtime.h"
#include ""


 namespace Cuda_Runtime
 {
 	__global__ void compute(const bool *primary_device_buffer, bool *secondary_device_buffer, int world_width)
 	{
        int index = blockIdx.x * blockDim.x +threadIdx.x;
 		const int x_cord = index % world_width;
 		const int y_cord = index / world_width;
 		int alive_neighbours = 0;
        for (int k = -1; k < 2; k++)
        {
            for (int j = -1; j < 2; j++)
            {
                if (j == 0 && k == 0)
                {
                    continue;
                }
                if (primary_device_buffer[(y_cord + k) * world_width + (x_cord + j)] == true)
                {
                    alive_neighbours++;
                }
            }
        }
 		if(alive_neighbours < 2 || alive_neighbours > 3)
 		{
 			secondary_device_buffer[index] = false;
 		}
 		else if(alive_neighbours == 3)
 		{
 			secondary_device_buffer[index] = true;
 		}
 		else
 		{
 			secondary_device_buffer[index] = primary_device_buffer[index];
 		}
 	}

 	ComputeGPU::ComputeGPU(int world_width, bool *start_world_frame)
 	{
 		gpu_state_ = INIT;
 		this->world_width = world_width;
 		this->host_world_buffer = start_world_frame;
 	}

 	void ComputeGPU::init()
 	{
 		buffer_size = world_width * world_width * sizeof(bool);
 		//GPU device memory allocation
 		hipMalloc(&primary_device_world_buffer, buffer_size);
 		hipMalloc(&secondary_device_world_buffer, buffer_size);

 		//Copy world buffer to GPU device
 		hipMemcpy(primary_device_world_buffer, host_world_buffer, buffer_size,hipMemcpyHostToDevice);

        gpu_state_ = RUNNING;
 	}
 	
 	void ComputeGPU::run()
 	{
 		while(gpu_state_ == RUNNING)
 		{
 			compute <<< world_width, world_width >>>(primary_device_world_buffer, secondary_device_world_buffer, world_width);
 			hipMemcpy(host_world_buffer, secondary_device_world_buffer, buffer_size,hipMemcpyDeviceToHost);
 			hipMemcpy(primary_device_world_buffer, secondary_device_world_buffer, buffer_size,hipMemcpyDeviceToDevice);
 			GRID->update_world(host_world_buffer);
 			GRID->print();
 		}
 	}

 	void ComputeGPU::shutdown()
 	{
 		free(host_world_buffer);
 		hipFree(primary_device_world_buffer);
 		hipFree(secondary_device_world_buffer);
        gpu_state_ = SHUTDOWN;
 	}
}	