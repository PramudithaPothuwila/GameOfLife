#include "hip/hip_runtime.h"
#include "CPUComputeMultiThread.cuh"
#include <thread>
#include <iostream>


namespace GameOfLife
{
	void CPUComputeMultiThread::compute(grid *world)
	{
		
	}

	void CPUComputeMultiThread::computeCell(grid *worldSubSector, int x, int y)
	{
		
	}

	void CPUComputeMultiThread::worldSlicer(const grid *world, const std::vector<grid> *subSectors, int subSectorCount)
	{
		const int size = world->size();
		for(int outerX = 2; outerX < size; outerX + 3)
		{
			for(int outerY = 2; outerY < size; outerY + 3)
			{
				grid subSector;
				for(int innerX = -2; innerX < 3; innerX++)
				{
					std::vector<int> row;
					for(int innerY = -2; innerY < 3; innerY++)
					{
						switch(world->at(outerX + innerX).at(outerY + innerY))
						{
							case 0:
								row.push_back(0);
								break;
							case 1:
								row.push_back(1);
								break;
							default:
								row.push_back(0);
								break;
						}
					}
					subSector.push_back(row);
				}
				//subSectors->push_back(subSector);
			}
		}
	}

	void CPUComputeMultiThread::threadManager(const grid *world)
	{
		const unsigned int processor_count = std::thread::hardware_concurrency();
		const int subSectorCount = std::ceil(static_cast<double>(world->size() * world->size())/9);
		std::vector<grid> subSectors;
		worldSlicer(world, &subSectors, subSectorCount);

	}
}