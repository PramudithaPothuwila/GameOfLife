#include "hip/hip_runtime.h"
#include "WorldGrid.cuh"
#include <cmath>

namespace GameOfLife
{
	WorldGrid::WorldGrid(int width)
	{
		grid = new int[width * width];
		int sectorCount = pow(ceil(static_cast<double>(gridWidth) / 5),2);
	}

	WorldGrid::~WorldGrid()
	{
		delete[] grid;
	}

	void WorldGrid::setCell(const int x,const int y,const int value)
	{
		grid[x * gridWidth + y] = value;
	}

	int WorldGrid::getCell(const int x,const int y)
	{
		return grid[x * gridWidth + y];
	}

	int WorldGrid::getSectorCount()
	{
		return pow(ceil(static_cast<double>(gridWidth) / 5),2);
	}

	int WorldGrid::getWorldWidth()
	{
		return gridWidth;
	}
	
	SuperSector::SuperSector(const int x,const int y, WorldGrid world)
	{
		for (int i = 0; i < 5; i++)
		{
			for (int j = 0; j < 5; j++)
			{
				this->superSector[x+y*5] = world.getCell(x + i, y + j);
			}
		}
		
	}

	SuperSector::~SuperSector()
	{
		delete[] superSector;
	}
	
	int SuperSector::getCellState(const int &x,const int &y) const
	{
		return superSector[y*5+x];
	}

	void SuperSector::setCellState(const int& x, const int& y,const int state)
	{
		superSector[y*5+x] = state;
	}
	
	
	Sector::Sector(const int x,const int y) : SuperSector(x,y)
	{
		
	}
	
	int Sector::getNeighbors(const int& x, const int& y) const
	{
		int neighbors = 0;
		for(int i = 0; i < 9; i++)
		{
			if(i ==4)
				continue;
			neighbors = neighbors + getCellState(x + i % 3, y + i / 3);
		}
		return 0;
	}

	void Sector::updateCellState(const int x,const int y)
	{
		const int neighbors = getNeighbors(x, y);
		if(getCellState(x, y) == 1)
		{
			if(neighbors < 2 || neighbors > 3)
			{
				setCellState(x, y, 0);
			}
		}
		else
		{
			if(neighbors == 3)
			{
				setCellState(x, y, 1);
			}
		}
	}

	void Sector::computeSector()
	{
		for(int i = 0; i < 9; i++)
		{
			updateCellState(i % 3, i / 3);
		}
	}
}