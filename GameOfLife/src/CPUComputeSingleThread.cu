﻿#include "CPUComputeSingleThread.cuh"
#include <thread>

namespace GameOfLife
{
	void CPUComputeSingleThread::Compute(std::vector<std::vector<int>>  *world)
	{
		while(true)
		{
			computeNextGeneration(world);
		}
	}

	int CPUComputeSingleThread::getNeighbours(const std::vector<std::vector<int>> *world, int x, int y)
	{
		int neighbours = 0;

		for (int i = -1; i < 2; i++)
		{
			for (int j = -1; j < 2; j++)
			{
				if (i == 0 && j == 0)
					continue;

				if (x + i < 0 || x + i >= world->size())
					continue;

				if (y + j < 0 || y + j >= world->at(0).size())
					continue;

				if (world->at(x + i).at(y + j) == 1)
					neighbours++;
			}
		}
		return neighbours;
	}

	void CPUComputeSingleThread::computeNextGeneration(std::vector<std::vector<int>> *world)
	{
		const auto newWorld = new std::vector<std::vector<int>>(world->size(), std::vector<int>(world->size(), 0));
		for(int i = 0; i < world->size(); i++)
		{
			for(int j = 0; j < world->size(); j++)
			{
				const int neighbours = getNeighbours(world, i, j);
				if(neighbours < 2 || neighbours > 3)
				{
					(*newWorld)[i][j] = 0;
				}
				else if(neighbours == 3)
				{
					(*newWorld)[i][j] = 1;
				}
				else
				{
					(*newWorld)[i][j] = (*world)[i][j];
				}
			}
		}
		free(world);
		world = newWorld;
	}
}