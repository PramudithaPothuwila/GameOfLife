﻿#include "Application.cuh"
#include "ImageProcessing.cuh"
#include <iostream>
#include <vector>
#include <opencv2/core/utils/logger.hpp>


std::vector<std::vector<int>> world; // 2D vector to store the world

void Application::Application::init()
{
    setLogLevel(cv::utils::logging::LogLevel::LOG_LEVEL_SILENT); // Disable OpenCV logging
    int grid;
    std::cout << "Initializing application..." << std::endl;
    std::cout << "Enter the Grid: "; // Enter the grid size
    std::cin >> grid;
    world.reserve(grid); // Reserve the size of the grid
    std::cout << "If you want to enter the world manually, press 1. If you want to generate the world by image, press 2." << std::endl;
}

int Application::Application::run()
{
    ImageProcessing::LoadImage("C:\\Users\\Buddy365\\Pictures\\Me\\IMG_20211206_001341.jpg");
    return 0;
}

void Application::Application::shutdown()
{
    
}
